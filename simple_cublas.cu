#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hipblas.h>

void vector_init(float *a, int n){
    for (int i = 0; i < n; i++){
        a[i] = (float)(rand() % 100);
    }
}

void check_answer(float *a, float *b, float *c, float factor, int n){
    for (int i = 0; i < n; i++){
        assert(c[i] == factor * a[i] + b[i]);
    }
}

int main(){
    int n = 1 << 16;

    size_t bytes = n*sizeof(float);

    float *h_a, *h_b, *h_c;
    float *d_a, *d_b;

    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);


    vector_init(h_a, n);
    vector_init(h_b, n);
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    hipblasSetVector(n, sizeof(float), h_a, 1, d_a, 1);
    hipblasSetVector(n, sizeof(float), h_b, 1, d_b, 1);
    
    const float scale = 2.0f;
    hipblasSaxpy(handle, n, &scale, d_a, 1, d_b, 1);

    hipblasGetVector(n, sizeof(float), d_b, 1, h_c, 1);
   
	check_answer(h_a, h_b, h_c, scale, n);

	hipblasDestroy(handle);

	hipFree(d_a);
	hipFree(d_b);
	free(h_a);
	free(h_b);

	return 0;

}