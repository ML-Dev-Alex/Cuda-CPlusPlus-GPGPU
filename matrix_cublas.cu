#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>


void check_answer(float *a, float *b, float *c, int n){
    float temp;
    float epsilon = 0.001;

    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            temp = 0;
            for (int k = 0; k < n; k++){
                temp += a[k * n + i] * b[j * n + k];
            }
            assert(fabs(c[j * n + i] - temp) < epsilon);
        }
    }
}

int main(){
    int n = 1 << 10;

    size_t bytes = n * n * sizeof(float);

    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;

    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)123);

    hiprandGenerateUniform(prng, d_a, n*n);
    hiprandGenerateUniform(prng, d_b, n*n);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    float alpha = 1.0f;
    float beta = 0.0f;

    // c = (alpha*a) * b + (beta*c)
    // (m X n) * (n * k) = (m X k)
    //          handle, operation,   operation,   m, n, k, alpha,   A, lda, B, ldb, beta,  C, ldc
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_a, n, d_b, n, &beta, d_c, n);

    hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
   
	check_answer(h_a, h_b, h_c, n);

	hipblasDestroy(handle);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	free(h_a);
	free(h_b);
	free(h_c);

	return 0;

}