#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>

__global__ void vectorAdd(int* a, int* b, int* c, int n){
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (tid < n){
        c[tid] = a[tid] + b[tid];
    }

}

void matrix_init(int* a, int n){
    for (int i = 0; i < n; i++){
        a[i] = rand()%100;
    }
}

void check_error(int* a, int* b, int* c, int n){
    for (int i = 0; i < n; i++){
        assert(c[i] == a[i] + b[i]);
    }
}

int main(){
    // 2^16
    int n = 1 << 16;

    // h_ = host variables (cpu)
    int *h_a, *h_b, *h_c;

    // device variables (gpu)
    int *d_a, *d_b, *d_c;

    size_t bytes = sizeof(int) * n;

    // Allocate host memory
    h_a = (int*)malloc(bytes);
    h_b = (int*)malloc(bytes);
    h_c = (int*)malloc(bytes);

    // Allocate device memory
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Initialize vectors with random values
    matrix_init(h_a, n);
    matrix_init(h_b, n);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    int NUM_THREADS = 256;

    int NUM_BLOCKS = (int)ceil(n / NUM_THREADS);

    vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, n);

    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    check_error(h_a, h_b, h_c, n);

    printf("Completed.\n");

    return 0;


}